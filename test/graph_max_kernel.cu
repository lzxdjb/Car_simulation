#include "hip/hip_runtime.h"
#include <torch/extension.h>
using namespace at;
// sdsd
__device__ float atomicMaxFloat(float* address, float val) {
    int* address_as_i = (int*)address;
    int old = *address_as_i, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed, __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);

    return __int_as_float(old);
}

__global__ void graph_max_kernel(const float* data, float* max_val, int64_t num_nodes) {
    extern __shared__ float shared_data[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < num_nodes) {
        shared_data[tid] = data[i];
    } else {
        shared_data[tid] = -FLT_MAX;
    }

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data[tid] = fmaxf(shared_data[tid], shared_data[tid + s]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicMaxFloat(max_val, shared_data[0]);
    }
}

at::Tensor graph_max_cuda(torch::Tensor data, torch ::Tensor max_val) {
    const int threads = 1024;
    const int blocks = (data.size(0) + threads - 1) / threads;
    const int shared_memory = threads * sizeof(float);

    graph_max_kernel<<<blocks, threads, shared_memory>>>(data.data_ptr<float>(), max_val.data_ptr<float>(), data.size(0));
}
